#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <iostream>
#include <memory>
#include <string>


/*
	Tutaj mamy testową klasę, której zadaniem jest trzymanie listy Int ów

*/
class Entity {
public:
	int* data;
	Entity(int size) : data(new int[size]) {}
	~Entity() { delete[] data; }
};



/*
	Funkcja globalna przechodzimy przez każdy element listy z klasy entity i mnożymy go przez 2
    Jest to funkcja CUDA która wykonuje tą operacje na GPU

*/
__global__ void kernel(int* deviceData, int size) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < size) {
        deviceData[idx] *= 2;
        printf("Thread %d: data[%d] = %d\n", idx, idx, deviceData[idx]); /* Tutaj też print żeby było widać bez debuggera co i jak */
    }
}

/* Tutaj mamy dwie podstawowe funkcje to przekazywania danych, od hosta(CPU) do GPU i w drugą stronę*/
void copyToGPU(int* device_ptr, int* host_ptr, int size) {
    hipMemcpy(device_ptr, host_ptr, size * sizeof(int), hipMemcpyHostToDevice);
}

/* Tutaj trzeba pamiętać aby przekazać sahred_ptr bo jeśli zmodyfikujemy dane przekazane z Entity to faktycznie Kernel wykona obliczenia, ale te dane są alokowane w innym miejscu w pamięci i faktycznie nie modyfikują
danych w sharedEntity
*/
void copyFromGPUToHost(int* device_ptr, std::shared_ptr<Entity> sharedEntity, int size) {
    hipMemcpy(sharedEntity->data, device_ptr, size * sizeof(int), hipMemcpyDeviceToHost);
}

int main() {
    int size = 1024;

    {
        std::shared_ptr<Entity> e0;
        {
            std::shared_ptr<Entity> sharedEntity = std::make_shared<Entity>(size);
            e0 = sharedEntity;

            
            for (int i = 0; i < size; i++) {
                sharedEntity->data[i] = i;
            }

            int* deviceData;
            hipMalloc(&deviceData, size * sizeof(int));

            copyToGPU(deviceData, sharedEntity->data, size); 
            /* 
            this	0x000000a8f178f628 shared_ptr {data=0x000001dbec1e9af0 {0} } [2 strong refs] [make_shared]	const std::shared_ptr<Entity> *
            */

            
            /* Tutaj wiem, że podkresla tą jedną strzałkę, ale nie wiem czym to jest spowodowane, nie wygląda jakby powodowało jakieś błędy*/
            int threadsPerBlock = 256; 
            int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock; /* Tutaj mi chat podpowiedział jak to wykalkulować jak coś*/

            kernel <<<blocksPerGrid, threadsPerBlock >> > (deviceData, size); /* Nie przejmuj się tym podkreśleniem, jest to chyba wina interpretera, bo w rzeczywistośc
            nie jest to błąd */
            hipDeviceSynchronize();
            copyFromGPUToHost(deviceData, sharedEntity, size);

            hipFree(deviceData); /* Zwalniamy całą używaną pamięć zarezerwowaną dla naszego deviceData */

            /* Tutaj dodałem pętle for by printować te dane, jeśli nie chcesz zaglądać w debuggera */
            std::cout << "Modified Data:" << std::endl;
            for (int i = 0; i < size; i++) {
                std::cout << sharedEntity->data[i] << " ";
            }
        }
    }

    std::cin.get();
    return 0;
}